#include "hip/hip_runtime.h"
#include "IMGA.cuh"

int main()
{
	//-------------- Problem variables ----------------
	// number of agents
	int numAgents = 0;
	// number of schedules
	int numSchedules = 0;
	// number of periods
	int numPeriods = 0;
	// length of read_arrE
	int lenArrE = 0;
	// array of count of schedules per Agent
	int *arrASchCount = NULL;
	// array of cummulative count of schedules per Agent
	int *arrAScanSchCount = NULL;
	// length of arrAScanSchCount
	int lenArrL = 0;
	// array of schedules index per Agent => try it on constant memory
	int *arrL = NULL;
	// array of schedules index per Agent host version
	int *read_arrE = NULL;
	// array of schedules index per Agent device version
	int *arrE;
	// array of number of Agents required per period
	int *arrN = NULL;
	// device id
	int deviceId = hipGetDevice(&deviceId);
	// hiprand state
	hiprandState *d_state;

	// ----------------- Genetic variables ------------------

	// initilize global variables
	initGblVars(numAgents, numSchedules, numPeriods);
	// initilize arrays
	arrASchCount = (int *)malloc(sizeof(int) * numAgents);
	arrAScanSchCount = (int *)malloc(sizeof(int) * numAgents);

	// read csv data A.csv
	readCSV_A(numAgents, arrASchCount, arrAScanSchCount, lenArrL);
	// read csv data L.csv+
	arrL = (int *)malloc(sizeof(int) * lenArrL);
	readCSV_L(arrL, lenArrL);

	// read csv data E.csv
	lenArrE = numSchedules * numPeriods;
	read_arrE = (int *)malloc(sizeof(int) * lenArrE);
	readCSV_E(read_arrE, lenArrE);
	// check schedule
	// for (int p = 0; p < numPeriods; p++)
	//	printf("period %i: %i\n", p+1, read_arrE[1774*numPeriods + p]);

	// read csv data P.csv
	arrN = (int *)malloc(sizeof(int) * numPeriods);
	readCSV_P(arrN, numPeriods);

	// unified memory for arrE
	hipMallocManaged(&arrE, lenArrE * sizeof(int));
	// hint to prioritize host transfer
	hipMemAdvise(arrE, lenArrE * sizeof(int), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
	// read data
	for (int e = 0; e < lenArrE; e++)
	{
		arrE[e] = read_arrE[e];
	}

	// hint for read mostly global data
	hipMemAdvise(arrE, lenArrE * sizeof(int), hipMemAdviseSetReadMostly, deviceId);
	// prefetching from host to device
	hipMemPrefetchAsync(arrE, lenArrE * sizeof(int), deviceId);

	// hiprand memory allocation
	hipMallocManaged(&d_state, BLOCKS_PER_GRID * THREADS_PER_BLOCK * sizeof(hiprandState));
	hipMemAdvise(d_state, BLOCKS_PER_GRID * THREADS_PER_BLOCK * sizeof(hiprandState), hipMemAdviseSetPreferredLocation, deviceId);
	setup_curand<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_state);
	hipDeviceSynchronize();
	hipMemAdvise(d_state, BLOCKS_PER_GRID * THREADS_PER_BLOCK * sizeof(hiprandState), hipMemAdviseSetReadMostly, deviceId);

	// allocate Device constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(const_numAgents), &numAgents, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_numSchedules), &numSchedules, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_numPeriods), &numPeriods, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_lenArrL), &lenArrL, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_arrASchCount), arrASchCount, numAgents * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_arrAScanSchCount), arrAScanSchCount, numAgents * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_arrL), arrL, lenArrL * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_arrN), arrN, lenArrE * sizeof(int));

	// execute kernel
	printf("\nblocks: %i", BLOCKS_PER_GRID);
	printf("\nthreads: %i", THREADS_PER_BLOCK);
	size_t shared_bytes = SUB_POPULATION_SIZE * numAgents * sizeof(int);
	printf("\nshared_bytes: %zu bytes", shared_bytes);
	kernel_IMGA<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK, shared_bytes, 0>>>(arrE, d_state);
	hipDeviceSynchronize();
	// deallocate dynamic memory
	free(arrASchCount);
	free(arrAScanSchCount);
	free(arrL);
	free(read_arrE);
	free(arrN);
	hipFree(arrE);
	hipFree(d_state);
	// reset device
	hipDeviceReset();
}

__host__ void initGblVars(int &numAgents, int &numSchedules, int &numPeriods)
{
	// initilize gbl variables
	numAgents = countRows("../MILP/A.csv") - 1;
	printf("\nnum agents: %i", numAgents);
	numSchedules = countRows("../MILP/S.csv") - 1;
	printf("\nnum schedules: %i", numSchedules);
	numPeriods = countRows("../MILP/P.csv") - 1;
	printf("\nnum periods: %i\n", numPeriods);
}

__host__ int countRows(string filePath)
{
	// count rows in a csv file
	int rows = 0;
	ifstream file;
	file.open(filePath);
	string line;
	while (getline(file, line))
		rows++;
	file.close();
	return rows;
}

__host__ void readCSV_A(int &numAgents, int *arrASchCount, int *arrAScanSchCount, int &lenArrL)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/A.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < numAgents) // while the end of file is NOT reached
		{
			arrAScanSchCount[i] = lenArrL;
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			arrASchCount[i] = stoi(col);
			lenArrL += arrASchCount[i];
			getline(coeff, col, '\n');
			// printf("\nagent %i: %i\n", i, arrASchCount[i]);
			// printf("agent scan %i: %i\n", i, arrAScanSchCount[i]);
			// printf("agent total scan: %i\n", lenArrL);
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_L(int *arrL, int &lenArrL)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/L.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < lenArrL) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			arrL[i] = stoi(col);
			getline(coeff, col, '\n');
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_E(int *read_arrE, int &lenArrE)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/E.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < lenArrE) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			read_arrE[i] = stoi(col);
			// printf("\nE %i: %i", i, read_arrE[i]);
			getline(coeff, col, '\n');
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_P(int *arrN, int &numPeriods)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/P.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < numPeriods) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			getline(coeff, col, '\n');
			arrN[i] = stoi(col);
			// printf("\nperiod %i: %i", i, arrN[i]);
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__global__ void setup_curand(hiprandState *state)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(blockIdx.x, threadIdx.x, 0, &state[tid]);
}

__global__ void kernel_IMGA(int *arrE, hiprandState *state)
{
	// initlize cooperative groups
	// the grid represents the global population
	cg::grid_group grid = cg::this_grid();
	// each block represents an island population
	cg::thread_block block = cg::this_thread_block();
	// each tile represents an individual
	cg::thread_block_tile<THREADS_PER_INDIVIDUAL> tile_individual = cg::tiled_partition<THREADS_PER_INDIVIDUAL>(block);

	// initilize population
	extern int __shared__ subPopulation[];

	// Copy random number state to local memory (registers) for efficiency
	hiprandState localState = state[grid.thread_rank()];

	for (int a = tile_individual.thread_rank(); a < const_numAgents; a += tile_individual.size())
	{
		int random_value = hiprand_uniform(&localState)*const_arrASchCount[a];
		subPopulation[tile_individual.meta_group_rank()*const_numAgents + a] = const_arrL[const_arrAScanSchCount[a] + random_value];
	}
	cg::sync(block);
}