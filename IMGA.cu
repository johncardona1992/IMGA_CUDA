#include "hip/hip_runtime.h"
#include "IMGA.cuh"

int main()
{
	hipError_t err = hipSuccess;
	//-------------- Problem variables ----------------
	// number of agents
	int numAgents = 0;
	// number of schedules
	int numSchedules = 0;
	// number of periods
	int numPeriods = 0;
	// agents IDs
	vector<string> agentsIDS;
	// Schedules IDs
	vector<string> schedulesIDS;
	// length of read_arrE
	int lenArrE = 0;
	// array of count of schedules per Agent
	int *arrASchCount = NULL;
	// array of cummulative count of schedules per Agent
	int *arrAScanSchCount = NULL;
	// length of arrAScanSchCount
	int lenArrL = 0;
	// array of schedules index per Agent => try it on constant memory
	int *arrL = NULL;
	// array of schedules index per Agent host version
	int *read_arrE = NULL;
	// array of schedules index per Agent device version
	int *arrE;
	// array of number of Agents required per period
	int *arrN = NULL;
	// device id
	int deviceId = hipGetDevice(&deviceId);
	// hiprand state
	hiprandState *d_state;
	// vector of emigrants chromosomes
	int *emigrants;
	// vector of fitness from emigrants
	int *fitness_emigrants;
	// best island solution
	int *global_solution;
	// best island solution
	int *islands_fitness;
	// best fitness solution
	int *best_fitness;

	// ----------------- Genetic variables ------------------

	// initilize global variables
	initGblVars(numAgents, numSchedules, numPeriods);
	// initilize arrays
	arrASchCount = (int *)malloc(sizeof(int) * numAgents);
	arrAScanSchCount = (int *)malloc(sizeof(int) * numAgents);

	// read csv data A.csv
	readCSV_A(numAgents, arrASchCount, arrAScanSchCount, lenArrL, agentsIDS);
	// read csv data S.csv
	readCSV_S(numSchedules, schedulesIDS);
	// read csv data L.csv+
	arrL = (int *)malloc(sizeof(int) * lenArrL);
	readCSV_L(arrL, lenArrL);
	printf("len L: %i", lenArrL);
	// read csv data E.csv
	lenArrE = numSchedules * numPeriods;
	read_arrE = (int *)malloc(sizeof(int) * lenArrE);
	readCSV_E(read_arrE, lenArrE);
	// check schedule
	// for (int p = 0; p < numPeriods; p++)
	//	printf("period %i: %i\n", p+1, read_arrE[1774*numPeriods + p]);

	// read csv data P.csv
	arrN = (int *)malloc(sizeof(int) * numPeriods);
	readCSV_P(arrN, numPeriods);

	// unified memory for arrE
	err = hipMallocManaged(&arrE, lenArrE * sizeof(int));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector arrE (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// hint to prioritize host transfer
	hipMemAdvise(arrE, lenArrE * sizeof(int), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
	// read data
	for (int e = 0; e < lenArrE; e++)
	{
		arrE[e] = read_arrE[e];
	}

	// hiprand memory allocation
	err = hipMallocManaged(&d_state, BLOCKS_PER_GRID * THREADS_PER_BLOCK * sizeof(hiprandState));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_state (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// emigrants memory allocation
	err = hipMallocManaged(&emigrants, BLOCKS_PER_GRID * MIGRATION_SIZE * AGENTS_SIZE * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector emigrants (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// fitness emigrants memory allocation
	err = hipMallocManaged(&fitness_emigrants, BLOCKS_PER_GRID * MIGRATION_SIZE * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector fitness_emigrants (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// global solutions memory allocation
	err = hipMallocManaged(&global_solution, AGENTS_SIZE * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector global_solution (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// global solutions memory allocation
	err = hipMallocManaged(&islands_fitness, BLOCKS_PER_GRID * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector islands_fitness (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// best fitness solution memory allocation
	err = hipMallocManaged(&best_fitness, sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector best_fitness (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// launch init kernel
	hipMemAdvise(d_state, BLOCKS_PER_GRID * THREADS_PER_BLOCK * sizeof(hiprandState), hipMemAdviseSetPreferredLocation, deviceId);
	setup_curand<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_state);
	hipDeviceSynchronize();

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch setup_curand kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//------- allocate Device constant memory-----
	err = hipMemcpyToSymbol(HIP_SYMBOL(const_numSchedules), &numSchedules, sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_numSchedules (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(const_numPeriods), &numPeriods, sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_numPeriods (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(const_lenArrL), &lenArrL, sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_lenArrL (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//------- allocate Device constant arrays memory-----
	err = hipMemcpyToSymbol(HIP_SYMBOL(const_arrASchCount), arrASchCount, numAgents * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_arrASchCount (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(const_arrAScanSchCount), arrAScanSchCount, numAgents * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_arrAScanSchCount (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(const_arrL), arrL, lenArrL * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_arrL (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(const_arrN), arrN, numPeriods * sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device constant const_arrN (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//-----hints------ unified memory
	// hint for read mostly global data
	hipMemAdvise(arrE, lenArrE * sizeof(int), hipMemAdviseSetReadMostly, deviceId);
	// prefetching from host to device
	hipMemPrefetchAsync(arrE, lenArrE * sizeof(int), deviceId);
	hipMemAdvise(d_state, BLOCKS_PER_GRID * THREADS_PER_BLOCK * sizeof(hiprandState), hipMemAdviseSetReadMostly, deviceId);
	hipMemAdvise(emigrants, BLOCKS_PER_GRID * MIGRATION_SIZE * AGENTS_SIZE * sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);
	hipMemAdvise(fitness_emigrants, BLOCKS_PER_GRID * MIGRATION_SIZE * sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);
	hipMemAdvise(global_solution, AGENTS_SIZE * sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);
	hipMemAdvise(islands_fitness, BLOCKS_PER_GRID * sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);
	hipMemAdvise(best_fitness, sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);

	//------------------- calculate theorical occupancy -------------------
	int dev = 0;
	int supportsCoopLaunch = 0;
	hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
	if (!supportsCoopLaunch)
	{
		printf(
			"\nSelected GPU (%d) does not support Cooperative Kernel Launch, "
			"Waiving the run\n",
			dev);
		exit(0);
	}

	/// This will launch a grid that can maximally fill the GPU, on the default stream with kernel arguments
	int numBlocksPerSm = 0;
	// Number of threads my_kernel will be launched with
	int numThreads = THREADS_PER_BLOCK;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, kernel_IMGA, numThreads, 0);
	// launch
	void *kernelArgs[] = {
		(void *)&arrE,
		(void *)&d_state,
		(void *)&emigrants,
		(void *)&fitness_emigrants,
		(void *)&global_solution,
		(void *)&islands_fitness,
		(void *)&best_fitness,
	};
	dim3 dimBlock(numThreads, 1, 1);
	// dim3 dimGrid(deviceProp.multiProcessorCount * numBlocksPerSm, 1, 1);
	dim3 dimGrid(BLOCKS_PER_GRID, 1, 1);
	// execute kernel
	printf("\nblocks: %i", dimGrid.x);
	printf("\nthreads: %i", numThreads);
	hipLaunchCooperativeKernel((void *)kernel_IMGA, dimGrid, dimBlock, kernelArgs);
	// kernel_IMGA<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(arrE, d_state, emigrants, fitness_emigrants);
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel_IMGA kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// prefetch results from device to host
	hipMemPrefetchAsync(global_solution, AGENTS_SIZE * sizeof(int), hipCpuDeviceId);
	hipMemPrefetchAsync(best_fitness, sizeof(int), hipCpuDeviceId);
	printf("\n---------------\n");
	printf("\nbest fitness: %i\n", best_fitness[0]);
	for (int i = 0; i < AGENTS_SIZE; i++)
	{
		// printf("\nagent %i: sch %i", i, global_solution[i]);
		std::cout << "agent: " << agentsIDS[i] << " sch: "<< schedulesIDS[global_solution[i]]<< std::endl;
	}
	// export solution to csv
	printSolution(numAgents, agentsIDS, schedulesIDS, global_solution);
	// deallocate dynamic memory
	free(arrASchCount);
	free(arrAScanSchCount);
	free(arrL);
	free(read_arrE);
	free(arrN);
	hipFree(arrE);
	hipFree(d_state);
	hipFree(emigrants);
	hipFree(fitness_emigrants);
	hipFree(global_solution);
	hipFree(islands_fitness);
	hipFree(best_fitness);

	// reset device
	hipDeviceReset();
}

__host__ void printSolution(int &numAgents, vector<string> &agentsIDS, vector<string> &schedulesIDS, int *global_solution)
{
	char filename[] = "../MILP/result.csv";
	fstream appendFileToWorkWith;

	appendFileToWorkWith.open(filename, std::fstream::in | std::fstream::out | std::ofstream::trunc | std::fstream::app);

	// If file does not exist, Create new file
	if (!appendFileToWorkWith)
	{
		// cout << "Cannot open file, file does not exist. Creating new file..";

		appendFileToWorkWith.open(filename, fstream::in | fstream::out | fstream::trunc);
		appendFileToWorkWith << "A_ID,S_ID,x_value\n";
		for (int i = 0; i < numAgents; i++)
		{
			appendFileToWorkWith << agentsIDS[i] << "," << schedulesIDS[global_solution[i]] << ","
								 << "1"
								 << "\n";
		}

		appendFileToWorkWith.close();
	}
	else
	{ // use existing file

		appendFileToWorkWith << "A_ID,S_ID,x_value\n";
		for (int i = 0; i < numAgents; i++)
		{
			appendFileToWorkWith << agentsIDS[i] << "," << schedulesIDS[global_solution[i]] << ","
								 << "1"
								 << "\n";
		}
		appendFileToWorkWith.close();
	}
}

__host__ void initGblVars(int &numAgents, int &numSchedules, int &numPeriods)
{
	// initilize gbl variables
	numAgents = countRows("../MILP/A.csv") - 1;
	printf("\nnum agents: %i", numAgents);
	numSchedules = countRows("../MILP/S.csv") - 1;
	printf("\nnum schedules: %i", numSchedules);
	numPeriods = countRows("../MILP/P.csv") - 1;
	printf("\nnum periods: %i\n", numPeriods);
}

__host__ int countRows(string filePath)
{
	// count rows in a csv file
	int rows = 0;
	ifstream file;
	file.open(filePath);
	string line;
	while (getline(file, line))
		rows++;
	file.close();
	return rows;
}

__host__ void readCSV_A(int &numAgents, int *arrASchCount, int *arrAScanSchCount, int &lenArrL, vector<string> &agentsIDS)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/A.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < numAgents) // while the end of file is NOT reached
		{
			arrAScanSchCount[i] = lenArrL;
			getline(coeff, col, ',');
			agentsIDS.push_back(col);
			getline(coeff, col, ',');
			arrASchCount[i] = stoi(col);
			lenArrL += arrASchCount[i];
			getline(coeff, col, '\n');
			// printf("\nagent %i: %i\n", i, arrASchCount[i]);
			// printf("agent scan %i: %i\n", i, arrAScanSchCount[i]);
			// printf("agent total scan: %i\n", lenArrL);
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_S(int &numSchedules, vector<string> &schedulesIDS)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/S.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < numSchedules) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			schedulesIDS.push_back(col);
			getline(coeff, col, '\n');
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_L(int *arrL, int &lenArrL)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/L.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < lenArrL) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			arrL[i] = stoi(col);
			getline(coeff, col, '\n');
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_E(int *read_arrE, int &lenArrE)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/E.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < lenArrE) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			getline(coeff, col, ',');
			read_arrE[i] = stoi(col);
			// printf("\nE %i: %i", i, read_arrE[i]);
			getline(coeff, col, '\n');
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__host__ void readCSV_P(int *arrN, int &numPeriods)
{
	// read data A.csv
	string col; // variables from file are here
	// input filename
	string file = "../MILP/P.csv";

	// number of lines
	int i = 0;

	ifstream coeff(file); // opening the file.
	if (coeff.is_open())  // if the file is open
	{
		// ignore first line
		string line;
		getline(coeff, line);

		while (i < numPeriods) // while the end of file is NOT reached
		{
			getline(coeff, col, ',');
			getline(coeff, col, '\n');
			arrN[i] = stoi(col);
			// printf("\nperiod %i: %i", i, arrN[i]);
			i += 1; // increment number of lines
		}
		coeff.close(); // closing the file
	}
	else
		cout << "Unable to open file"; // if the file is not open output
}

__global__ void setup_curand(hiprandState *state)
{
	// each island has a different seed, and each individual has a different sequence
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(blockIdx.x, threadIdx.x, 0, &state[tid]);
}

__global__ void kernel_IMGA(int *arrE, hiprandState *state, int *emigrants, int *fitness_emigrants, int *global_solution, int *islands_fitness, int *best_fitness)
{
	// initlize cooperative groups
	// the grid represents the global population
	cg::grid_group grid = cooperative_groups::this_grid();
	// each block represents an island population
	cg::thread_block block = cg::this_thread_block();
	// each tile represents an individual
	cg::thread_block_tile<THREADS_PER_INDIVIDUAL> tile_individual = cg::tiled_partition<THREADS_PER_INDIVIDUAL>(block);
	// each tile represents a tournament
	cg::thread_block_tile<NUM_TOURNAMENTS> tile_tournament = cg::tiled_partition<NUM_TOURNAMENTS>(tile_individual);

	cg::sync(tile_individual);
	// shared memory
	// island population of parents
	int __shared__ subPopulation_source[SUBPOPULATION_BYTES];
	// island population of children
	int __shared__ subOffsprings_source[SUBPOPULATION_BYTES];
	// fitnes vector for each island
	int __shared__ arrFitness
		[SUB_POPULATION_SIZE];
	// Parent ID vector for each island
	int __shared__ arrParents[SUB_POPULATION_SIZE];
	// Highlander ID for each island
	int __shared__ highlander[1];
	// Highlander fitness for each island
	int __shared__ highlander_fitness[1];
	// Highlander Chromosome
	int __shared__ highlander_chromosome[AGENTS_SIZE];
	// Emigrant ID vector for each island
	int __shared__ arrEmigrantID[MIGRATION_SIZE];
	// weak ID vector for each island
	int __shared__ arrWeakID[MIGRATION_SIZE];

	int __shared__ *subPopulation;
	int __shared__ *subOffsprings;

	int __shared__ neighbor[1];

	if (block.thread_index().x == 0)
	{
		subPopulation = &subPopulation_source[0];
		subOffsprings = &subOffsprings_source[0];
		neighbor[0] = (block.group_index().x + 1);
		if (neighbor[0] >= BLOCKS_PER_GRID)
		{
			neighbor[0] = 0;
		}
	}
	cg::sync(block);
	// ------------------- Initilize sub-populations ------------------------------
	// Copy random number state to local memory (registers) for efficiency
	hiprandState localState = state[grid.thread_rank()];

	for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
	{
		float random_value = hiprand_uniform(&localState) * const_arrASchCount[a];
		int random_pos = (int)truncf(random_value);

		subPopulation[tile_individual.meta_group_rank() * AGENTS_SIZE + a] = const_arrL[const_arrAScanSchCount[a] + random_pos];
		//--------Validate initial Population
		// int idb = blockIdx.x;
		// if (idb == 0)
		// {
		// 	printf("\nblock: %i, individual: %i, agent: %i, feasible: %i, startID: %i, random: %i, scheduleID: %i", block.group_index().x, tile_individual.meta_group_rank(), a, const_arrASchCount[a], const_arrAScanSchCount[a], random_pos, const_arrL[const_arrAScanSchCount[a] + random_pos]);
		// }
	}

	//---------------------start epoch-----------------------------
	for (int epoch = 0; epoch < MAX_EPOCHES; epoch++)
	{
		//---------------------start generation------------------------
		for (int generation = 0; generation < MAX_GENERATIONS; generation++)
		{
			// initilize fitness and parent vectors for each generation
			if (tile_individual.thread_rank() == 0)
			{
				arrFitness[tile_individual.meta_group_rank()] = 0;
				arrParents[tile_individual.meta_group_rank()] = 0;
			}
			// syncronize all threads from the same island
			cg::sync(block);
			//------------------ calculate fitness--------------------------
			// local memory
			int objective = 0;
			int active_agents = 0;
			for (int p = 0; p < const_numPeriods; p++)
			{
				active_agents = 0;
				// grid stride loops along agents dimension
				for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
				{
					int idSchedule = subPopulation[tile_individual.meta_group_rank() * AGENTS_SIZE + a];
					active_agents += arrE[idSchedule * const_numPeriods + p];
					//  print schedules and set covering for period p
					// if (block.group_index().x == 0 && tile_individual.meta_group_rank() == 0)
					// {
					// 	printf("\nagent %i, schedule %i: %i", a, idSchedule, arrE[idSchedule * const_numPeriods + p]);
					// }
				}

				cg::sync(tile_individual);
				// reduce cooperative function
				active_agents = cg::reduce(tile_individual, active_agents, cg::plus<int>());

				// calculate objective funtion
				if (tile_individual.thread_rank() == 0)
				{
					// objective could be moved to shared memory
					objective = objective + max(const_arrN[p] - active_agents, 0);
					// print fo along the periods
					//  if (block.group_index().x == 0 && tile_individual.meta_group_rank() == 2)
					//  {
					//  	printf("\nPeriodo %i, Activos: %i, requeridos: %i, fo: %i", p, active_agents, const_arrN[p], objective);
					//  }
					// roulette selection
					// atomicAdd(&totalFitness[block.group_index().x],objective);
				}
			}
			if (tile_individual.thread_rank() == 0)
			{
				arrFitness[tile_individual.meta_group_rank()] = objective;
			}
			// print fitness vector for island 0
			// if (block.group_index().x == 27 && tile_individual.thread_rank() == 0)
			// {
			// 	printf("\nindividual %i: %i faltantes, %i", tile_individual.meta_group_rank(), objective,arrFitness[tile_individual.meta_group_rank()]);
			// }

			cg::sync(block);
			//-----------------------Elitism ------------------------------
			if (tile_individual.meta_group_rank() == 0)
			{
				int fitness = 1000000;
				for (int c = tile_individual.thread_rank(); c < SUB_POPULATION_SIZE; c += tile_individual.size())
				{
					fitness = min(arrFitness[c], fitness);
				}
				fitness = cg::reduce(tile_individual, fitness, cg::less<int>());
				for (int c = tile_individual.thread_rank(); c < SUB_POPULATION_SIZE; c += tile_individual.size())
				{
					if (tile_individual.shfl(fitness, 0) == arrFitness[c])
					{
						atomicExch(&highlander[0], c);
						atomicExch(&highlander_fitness[0], arrFitness[c]);
					}
				}
				cg::sync(tile_individual);
				// make a copy of highlander chromosome
				for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
				{
					highlander_chromosome[a] = subPopulation[highlander[0] * AGENTS_SIZE + a];
				}
			}
			cg::sync(block);
			// validate highlander
			// if (block.thread_index().x == 0 && block.group_index().x == 0)
			// {
			// 	for (int c = 0; c < SUB_POPULATION_SIZE; c++)
			// 	{
			// 		printf("\nindividual %i: %i", c, arrFitness[c]);
			// 	}
			// 	printf("\nbest %i",highlander[0]);
			// }
			//---------------- Migration -------------------------------
			if (generation == (MAX_GENERATIONS - 1))
			{
				if (tile_individual.meta_group_rank() < MIGRATION_SIZE)
				{
					// select emigrants
					int emigrantID = 0;
					float random_value = hiprand_uniform(&localState) * SUB_POPULATION_SIZE;
					emigrantID = (int)truncf(random_value);
					objective = arrFitness[emigrantID];
					cg::sync(tile_individual);
					objective = cg::reduce(tile_individual, objective, cg::less<int>());
					// fill up emigrants ID and respective fitness
					if (tile_individual.shfl(objective, 0) == arrFitness[emigrantID])
					{
						// shared memory - winners
						atomicExch(&arrEmigrantID[tile_individual.meta_group_rank()], emigrantID);
						// global memory - fitness of the winners
						atomicExch(&fitness_emigrants[block.group_index().x * MIGRATION_SIZE + tile_individual.meta_group_rank()], objective);
					}
					cg::sync(tile_individual);
					// select weak emigrants
					int weakID = 0;
					random_value = hiprand_uniform(&localState) * SUB_POPULATION_SIZE;
					weakID = (int)truncf(random_value);
					objective = arrFitness[weakID];
					cg::sync(tile_individual);
					objective = cg::reduce(tile_individual, objective, cg::greater<int>());
					// fill up weaks ID
					if (tile_individual.shfl(objective, 0) == arrFitness[weakID])
					{
						atomicExch(&arrWeakID[tile_individual.meta_group_rank()], weakID);
					}
					// copy emigrant chromosome from shared to global memory
					for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
					{
						emigrants[block.group_index().x * MIGRATION_SIZE * AGENTS_SIZE + tile_individual.meta_group_rank() * AGENTS_SIZE + a] = subPopulation[arrEmigrantID[tile_individual.meta_group_rank()] * AGENTS_SIZE + a];
					}
				}
				// a grid sync is necessary before starting migration from global to shared memory
				cg::sync(grid);
				if (tile_individual.meta_group_rank() < MIGRATION_SIZE)
				{
					for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
					{
						subPopulation[arrWeakID[tile_individual.meta_group_rank()] * AGENTS_SIZE + a] = emigrants[neighbor[0] * MIGRATION_SIZE * AGENTS_SIZE + tile_individual.meta_group_rank() * AGENTS_SIZE + a];
					}
					if (tile_individual.thread_rank() == 0)
					{
						arrFitness[arrWeakID[tile_individual.meta_group_rank()]] = fitness_emigrants[neighbor[0] * MIGRATION_SIZE + tile_individual.meta_group_rank()];
					}
				}
				// validate copy
				// if (block.group_index().x == 27 && block.thread_index().x == 0)
				// {
				// 	int k = 1;
				// 	for (int a = 0; a < AGENTS_SIZE; a++)
				// 	{
				// 		printf("\nagente %i: %i, %i", a, subPopulation[arrWeakID[k] * AGENTS_SIZE + a], emigrants[(neighbor[0] * MIGRATION_SIZE * AGENTS_SIZE) + (k * AGENTS_SIZE) + a]);
				// 	}
				// 	printf("\nneighbor %i", neighbor[0]);
				// 	printf("\nweakID %i", arrWeakID[k]);
				// 	printf("\nfitness %i, %i", arrFitness[arrWeakID[k]], fitness_emigrants[neighbor[0] * MIGRATION_SIZE + k]);
				// }
			}
			cg::sync(block);
			//---------end migration----------//
			//---------------- tournament selection --------------------
			if (tile_tournament.meta_group_rank() == 0)
			{

				int parentID = 0;
				float random_value = hiprand_uniform(&localState) * SUB_POPULATION_SIZE;
				parentID = (int)truncf(random_value);
				objective = arrFitness[parentID];
				cg::sync(tile_tournament);
				// get winner fitness by reduce cooperative function
				//  if (block.group_index().x == 27 && tile_tournament.meta_group_rank() == 0)
				//  {
				//   	printf("\nindividual %i: %i faltantes", tile_tournament.thread_rank(), objective);
				//  }
				objective = cg::reduce(tile_tournament, objective, cg::less<int>());
				// if (block.group_index().x == 0 && tile_tournament.meta_group_rank() == 0)
				// {
				//   	printf("\nindividual %i, parentID %i, fitness %i, minimo %i", tile_tournament.thread_rank(), parentID, arrFitness[parentID], tile_tournament.shfl(objective,0));
				// }
				// deterministic using atomic operators
				if (tile_tournament.shfl(objective, 0) == arrFitness[parentID])
				{
					atomicExch(&arrParents[tile_individual.meta_group_rank()], parentID);
				}
			}
			cg::sync(block);
			// if (block.group_index().x == 0 && tile_individual.meta_group_rank() == 0)
			// {
			//   	printf("\nparentID selected %i", arrParents[tile_individual.meta_group_rank()]);
			// }
			//-----------------------Crossover-------------------------
			// generate crossover point
			// first half from parent
			for (int a = tile_individual.thread_rank(); a < CROSSPOINT; a += tile_individual.size())
			{
				subOffsprings[tile_individual.meta_group_rank() * AGENTS_SIZE + a] = subPopulation[arrParents[tile_individual.meta_group_rank()] * AGENTS_SIZE + a];
			}
			// second half from individual
			for (int a = CROSSPOINT + tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
			{
				subOffsprings[tile_individual.meta_group_rank() * AGENTS_SIZE + a] = subPopulation[tile_individual.meta_group_rank() * AGENTS_SIZE + a];
			}
			cg::sync(block);
			// print crossover of one individual
			//  if (block.group_index().x == 0 && block.thread_rank() == 0)
			//  {
			//  	for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a++)
			//  	{
			//  		printf("\ngene %i, parent1: %i, parent2: %i, offspring: %i", a, subPopulation[1 * AGENTS_SIZE + a], subPopulation[arrParents[1] * AGENTS_SIZE + a], subOffsprings[1 * AGENTS_SIZE + a]);
			//  	}
			//  }
			//-----------------------Mutation-------------------------------
			for (int a = tile_individual.thread_rank(); a < AGENTS_SIZE; a += tile_individual.size())
			{
				float random_v = hiprand_uniform(&localState);
				if (random_v < MUTATION_RATE)
				{
					random_v = hiprand_uniform(&localState) * const_arrASchCount[a];
					int random_pos = (int)truncf(random_v);
					subOffsprings[tile_individual.meta_group_rank() * AGENTS_SIZE + a] = const_arrL[const_arrAScanSchCount[a] + random_pos];
					// if (block.group_index().x == 0)
					// 	printf("\nagent %i: idschedule %i", a, subOffsprings[tile_individual.meta_group_rank() * AGENTS_SIZE + a]);
				}
			}
			cg::sync(block);
			//-------------------------- replace old population -------------------
			// replace random child by highlander
			if (tile_individual.meta_group_rank() == 0)
			{
				int childID = 0;
				int random_value = hiprand_uniform(&localState) * SUB_POPULATION_SIZE;
				childID = (int)truncf(random_value);
				// not necesary to use sync(tile_individual)
				for (int c = tile_individual.thread_rank(); c < SUB_POPULATION_SIZE; c += tile_individual.size())
				{
					atomicExch(&subOffsprings[tile_individual.shfl(childID, 0) * AGENTS_SIZE + c], highlander_chromosome[c]);
				}
			}
			cg::sync(block);
			// validate replacement highlander
			// if (block.thread_index().x == 0 && block.group_index().x == 0)
			// {
			// 	for (int c = 0; c < AGENTS_SIZE; c++)
			// 	{
			// 		printf("\nposition %i: %i, %i", c, subPopulation[highlander[0] * AGENTS_SIZE + c], subOffsprings[0 * AGENTS_SIZE + c]);
			// 	}
			// }
			// children replace parents
			if (block.thread_index().x == 0)
			{
				int *p = &subPopulation[0];
				subPopulation = subOffsprings;
				subOffsprings = p;
			}

			// if (block.thread_index().x == 0)
			// {
			// 	printf("\nepoch %i, block %i, fitness %i", epoch, block.group_index().x, highlander_fitness[0]);
			// }
			// cg::sync(block);
		}
		//---------------------- end epoch ------------------------
	}
	cg::sync(block);
	// extract best solution
	if (block.thread_index().x == 0)
	{
		islands_fitness[block.group_index().x] = highlander_fitness[0];
	}
	cg::sync(grid);

	if (grid.thread_rank() == 0)
	{
		best_fitness[0] = 100000;
		for (int i = 0; i < BLOCKS_PER_GRID; i++)
		{
			best_fitness[0] = min(best_fitness[0], islands_fitness[i]);
			if (best_fitness[0] == islands_fitness[i])
			{
				emigrants[0] = i;
			}
		}
		// printf("\nbest %i, pos %i", best_fitness[0], emigrants[0]);
	}
	cg::sync(grid);
	if (block.group_index().x == emigrants[0] && block.thread_index().x == 0)
	{
		for (int a = 0; a < AGENTS_SIZE; a++)
		{
			global_solution[a] = highlander_chromosome[a];
			// printf("\nAgent %i: sch %i", a, highlander_chromosome[a]);
		}
	}
}